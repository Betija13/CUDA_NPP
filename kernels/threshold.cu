#include <npp.h>
#include <nppi.h>
#include <cstdio>
#include <hip/hip_runtime.h>

extern "C" void threshold(const unsigned char* d_input, unsigned char* d_output, int width, int height, int channels, unsigned char thresh) {
    // Ensure CUDA context is initialized
    hipFree(0);
    
    // Define the size of the region of interest (ROI)
    NppiSize oSizeROI = { width, height };
    
    // Define step sizes for input and output images
    int srcStep = width * channels;
    int dstStep = srcStep;
    
    // Status variable for NPP operations
    NppStatus status;
    
    if (channels == 1) {
        // Grayscale thresholding
        printf("--Applying grayscale thresholding with threshold value: %d\n", thresh);
        
        // Create a temporary buffer for the threshold operation
        unsigned char* d_temp = nullptr;
        hipError_t err = hipMalloc(&d_temp, width * height * sizeof(unsigned char));
        if (err != hipSuccess) {
            printf("hipMalloc failed for temporary buffer: %s\n", hipGetErrorString(err));
            return;
        }

        status = nppiThreshold_GTVal_8u_C1R(
            d_input, srcStep,
            d_output, dstStep,
            oSizeROI,
            thresh,     // Threshold value
            0           // Value for pixels below threshold
        );
        
        if (status != NPP_SUCCESS) {
            printf("nppiThreshold_GTVal_8u_C1R failed with status %d\n", status);
            hipFree(d_temp);
            return;
        }
        
        // Free temporary buffer
        hipFree(d_temp);
    }
    else if (channels == 3) {
        // RGB thresholding
        printf("--Applying RGB thresholding with threshold value: %d\n", thresh);
        
        // Create a temporary buffer for the threshold operation
        unsigned char* d_temp = nullptr;
        hipError_t err = hipMalloc(&d_temp, width * height * 3 * sizeof(unsigned char));
        if (err != hipSuccess) {
            printf("hipMalloc failed for temporary buffer: %s\n", hipGetErrorString(err));
            return;
        }
        
        
        // Create threshold array for RGB
        Npp8u threshArr[3] = { thresh, thresh, thresh };
        
        // Set all pixels in temp buffer to 255 (white) where input pixels >= threshold
        status = nppiThreshold_GTVal_8u_C3R(
            d_input, srcStep,
            d_temp, srcStep, 
            oSizeROI,
            threshArr,       // Threshold value for each channel
            threshArr        // Output value (255) for pixels above threshold
        );
        
        if (status != NPP_SUCCESS) {
            printf("nppiThreshold_GTVal_8u_C3R failed with status %d\n", status);
            hipFree(d_temp);
            return;
        }
        
        // Set all pixels in temp buffer to 0 (black) where input pixels < threshold
        Npp8u zeroArr[3] = { 0, 0, 0 };
        status = nppiThreshold_LTVal_8u_C3R(
            d_input, srcStep,
            d_temp, srcStep,
            oSizeROI,
            threshArr,       // Threshold value for each channel
            zeroArr          // Output value (0) for pixels below threshold
        );
        
        if (status != NPP_SUCCESS) {
            printf("nppiThreshold_LTVal_8u_C3R failed with status %d\n", status);
            hipFree(d_temp);
            return;
        }
        
        // Copy the result to output
        status = nppiCopy_8u_C3R(
            d_temp, srcStep,
            d_output, dstStep,
            oSizeROI
        );
        
        if (status != NPP_SUCCESS) {
            printf("nppiAnd_8u_C3IR failed with status %d\n", status);
            hipFree(d_temp);
            return;
        }
        
        // Free temporary buffer
        hipFree(d_temp);
    }
    else {
        printf("Error: Thresholding only supports 1 or 3 channel images.\n");
        return;
    }
    
    printf("--Thresholding completed successfully.\n");
}